#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <chrono>
#include "data.h"

__constant__ float const_tmin;
__constant__ float const_delta;
__constant__ int   const_coefficients;

__constant__ float const_pi;
__constant__ float const_pi_squared;
__constant__ float const_T;
__constant__ float const_pi_over_T;
__constant__ float constant_result_coefficient;

__global__ void fourier(float* results)
{
    __shared__ float shared_coefficients[MAX_COEFFICIENTS];

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    auto idx = threadIdx.x;

    for (auto k = idx; k < const_coefficients; k += blockDim.x)
    {
        auto denominator = 4.0f * (k + 1) * (k + 1) - 4.0f * (k + 1) + 1.0f;
        shared_coefficients[k] = 1.0f / denominator;
    }

    __syncthreads();

    if (tid >= gridDim.x * blockDim.x) return;

    auto t = const_tmin + tid * const_delta;
    auto sum = 0.0f;

    for (auto k = 1; k <= const_coefficients; ++k)
    {
        auto angle = (2 * k - 1) * const_pi_over_T * t;
        auto numerator = cosf(angle);
        sum += numerator * shared_coefficients[k - 1];
    }

    results[tid] = const_T * 0.5f - (constant_result_coefficient * sum);
}

void performColdRun(float tmin, float tmax, int length, int coefficients, float delta) {
    float* result_device;
    hipMalloc((void**)&result_device, length * sizeof(float));

    auto blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipMemcpyToSymbol(HIP_SYMBOL(const_tmin),   &tmin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_delta),  &delta, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_coefficients), &coefficients, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(const_pi), &pi, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_pi_squared), &pi_squared, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_T), &T, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_pi_over_T), &pi_over_T, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(constant_result_coefficient), &result_coefficient, sizeof(float));

    fourier<<<blocks, THREADS_PER_BLOCK>>>(result_device);
    hipDeviceSynchronize();

    float* result_host = new float[length];
    hipMemcpy(result_host, result_device, length * sizeof(float), hipMemcpyDeviceToHost);
    delete[] result_host;
    hipFree(result_device);
}

int main()
{

	printf("Performing cold run to warm up GPU...\n");
	performColdRun(tmin, tmax, length, coefficients, delta);
	printf("Cold run completed.\n\n");

	std::vector<double> prep_times, kernel_times, copy_times, delete_times;

	auto start_reps = std::chrono::high_resolution_clock::now();
	for (auto rep = 0; rep < NUM_REPS; ++rep) {
		auto prep_start = std::chrono::high_resolution_clock::now();
		
		float* result_device;
		hipMalloc((void**)&result_device, length * sizeof(float));

		auto blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		hipMemcpyToSymbol(HIP_SYMBOL(const_tmin),   &tmin, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(const_delta),  &delta, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(const_coefficients), &coefficients, sizeof(int));

		hipMemcpyToSymbol(HIP_SYMBOL(const_pi), &pi, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(const_pi_squared), &pi_squared, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(const_T), &T, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(const_pi_over_T), &pi_over_T, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(constant_result_coefficient), &result_coefficient, sizeof(float));

		auto prep_end = std::chrono::high_resolution_clock::now();
		prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());
		
		hipEvent_t kernel_start, kernel_stop;
		hipEventCreate(&kernel_start);
		hipEventCreate(&kernel_stop);
		hipEventRecord(kernel_start);
		
		fourier<<<blocks, THREADS_PER_BLOCK>>>(result_device);
		hipDeviceSynchronize();
		hipEventRecord(kernel_stop);
		hipEventSynchronize(kernel_stop);
		
		float kernel_ms = 0.0f;
		hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop);
		kernel_times.push_back(kernel_ms / 1000.0);
		
		hipEventDestroy(kernel_start);
		hipEventDestroy(kernel_stop);
		
		auto copy_start = std::chrono::high_resolution_clock::now();
		float* result_host = new float[length];
		hipMemcpy(result_host, result_device, length * sizeof(float), hipMemcpyDeviceToHost);
		auto copy_end = std::chrono::high_resolution_clock::now();
		copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());
		
		auto delete_start = std::chrono::high_resolution_clock::now();
		delete[] result_host;
		hipFree(result_device);
		auto delete_end = std::chrono::high_resolution_clock::now();
		
		delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
	}
	auto end_reps = std::chrono::high_resolution_clock::now();
	
	double prep_sum = 0, kernel_sum = 0, copy_sum = 0, delete_sum = 0;
	
	for (auto i = 0u; i < prep_times.size(); ++i) {
	    if(logReps) {
	        printf("Repetition %u:\n", i + 1);
        	printf("  Preparation time: %.6f s\n", prep_times[i]);
        	printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
        	printf("  Data copy time: %.6f s\n", copy_times[i]);
        	printf("  Memory deletion time: %.6f s\n", delete_times[i]);
	    }
		prep_sum += prep_times[i];
		kernel_sum += kernel_times[i];
		copy_sum += copy_times[i];
		delete_sum += delete_times[i];
	}
	
	auto n = static_cast<double>(prep_times.size());
	double prep_avg = prep_sum / n;
	double kernel_avg = kernel_sum / n;
	double copy_avg = copy_sum / n;
	double delete_avg = delete_sum / n;
	double prep_var = 0, kernel_var = 0, copy_var = 0, delete_var = 0;
	
	for (auto i = 0u; i < prep_times.size(); ++i) {
		prep_var += (prep_times[i] - prep_avg) * (prep_times[i] - prep_avg);
		kernel_var += (kernel_times[i] - kernel_avg) * (kernel_times[i] - kernel_avg);
		copy_var += (copy_times[i] - copy_avg) * (copy_times[i] - copy_avg);
		delete_var += (delete_times[i] - delete_avg) * (delete_times[i] - delete_avg);
	}
	
	double prep_std = std::sqrt(prep_var / n);
	double kernel_std = std::sqrt(kernel_var / n);
	double copy_std = std::sqrt(copy_var / n);
	double delete_std = std::sqrt(delete_var / n);
	
	printf("\nAverages over %zu repetitions:\n", prep_times.size());
	printf("  Avg preparation time: %.6f s (stddev: %.6f s)\n", prep_avg, prep_std);
	printf("  Avg kernel execution time: %.6f s (stddev: %.6f s)\n", kernel_avg, kernel_std);
	printf("  Avg data copy time: %.6f s (stddev: %.6f s)\n", copy_avg, copy_std);
	printf("  Avg memory deletion time: %.6f s (stddev: %.6f s)\n", delete_avg, delete_std);
	printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
	printf("=========================\n\n");
	return 0;
}