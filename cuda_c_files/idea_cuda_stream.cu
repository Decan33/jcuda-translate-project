#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <cstring>
#include "data.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void fourier(
    float tmin,
    float delta,
    int length,
    int coefficients,
    float pi,
    float pi_over_T,
    float result_coefficient,
    float T,
    float *results,
    int stream_offset,
    int stream_size)
{
    int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_local >= stream_size) return;
    int idx_global = idx_local + stream_offset;
    float t = tmin + idx_global * delta;
    float sum = 0.0f;
    for (int k = 1; k <= coefficients; ++k) {
        float angle = (2 * k - 1) * pi_over_T * t;
        float denominator = 4.0f * k * k - 4.0f * k + 1.0f;
        sum += cosf(angle) / denominator;
    }
    results[idx_local] = T * 0.5f - result_coefficient * sum;
}

void performColdRun(float tmin, float tmax, int length, int coefficients, float T, 
                   float delta, float pi, float pi_over_T, float result_coefficient) {
    int chunkSize = (length + NUM_STREAMS - 1) / NUM_STREAMS;
    hipStream_t streams[NUM_STREAMS];
    float* d_results[NUM_STREAMS];
    float* h_results[NUM_STREAMS];

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
        CUDA_CHECK(hipMalloc(&d_results[i], chunkSize * sizeof(float)));
        CUDA_CHECK(hipHostAlloc(&h_results[i], chunkSize * sizeof(float), hipHostMallocDefault));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int startIdx = i * chunkSize;
        int currentChunkSize = std::min(chunkSize, length - startIdx);
        int blocksPerGrid = (currentChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        fourier<<<blocksPerGrid, THREADS_PER_BLOCK, 0, streams[i]>>>(
            tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T,
            d_results[i], startIdx, currentChunkSize
        );
        CUDA_CHECK(hipGetLastError());
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int startIdx = i * chunkSize;
        int currentChunkSize = std::min(chunkSize, length - startIdx);
        CUDA_CHECK(hipMemcpy(h_results[i], d_results[i], currentChunkSize * sizeof(float), hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipFree(d_results[i]));
        CUDA_CHECK(hipHostFree(h_results[i]));
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
}

int main() {
    printf("Performing cold run to warm up GPU...\n");
    performColdRun(tmin, tmax, length, coefficients, T, delta, pi, pi_over_T, result_coefficient);
    printf("Cold run completed.\n\n");

    std::vector<double> prep_times, kernel_times, copy_times, delete_times;
    int chunkSize = (length + NUM_STREAMS - 1) / NUM_STREAMS;
    auto start_reps = std::chrono::high_resolution_clock::now();
    for (auto rep = 0; rep < NUM_REPS; ++rep) {
        auto prep_start = std::chrono::high_resolution_clock::now();
        hipStream_t streams[NUM_STREAMS];
        float* d_results[NUM_STREAMS];
        float* h_results[NUM_STREAMS];
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreate(&streams[i]));
            CUDA_CHECK(hipMalloc(&d_results[i], chunkSize * sizeof(float)));
            CUDA_CHECK(hipHostAlloc(&h_results[i], chunkSize * sizeof(float), hipHostMallocDefault));
        }
        auto prep_end = std::chrono::high_resolution_clock::now();
        prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());
        hipEvent_t kernel_start, kernel_stop;
        CUDA_CHECK(hipEventCreate(&kernel_start));
        CUDA_CHECK(hipEventCreate(&kernel_stop));
        CUDA_CHECK(hipEventRecord(kernel_start));
        for (int i = 0; i < NUM_STREAMS; ++i) {
            int startIdx = i * chunkSize;
            int currentChunkSize = std::min(chunkSize, length - startIdx);
            int blocksPerGrid = (currentChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            fourier<<<blocksPerGrid, THREADS_PER_BLOCK, 0, streams[i]>>>(
                tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T,
                d_results[i], startIdx, currentChunkSize
            );
            CUDA_CHECK(hipGetLastError());
        }
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }

        CUDA_CHECK(hipEventRecord(kernel_stop));
        CUDA_CHECK(hipEventSynchronize(kernel_stop));

        float kernel_ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop));
        kernel_times.push_back(kernel_ms / 1000.0);

        CUDA_CHECK(hipEventDestroy(kernel_start));
        CUDA_CHECK(hipEventDestroy(kernel_stop));
        auto copy_start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < NUM_STREAMS; ++i) {
            int startIdx = i * chunkSize;
            int currentChunkSize = std::min(chunkSize, length - startIdx);
            CUDA_CHECK(hipMemcpy(h_results[i], d_results[i], currentChunkSize * sizeof(float), hipMemcpyDeviceToHost));
        }

        auto copy_end = std::chrono::high_resolution_clock::now();
        copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());
        auto delete_start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipFree(d_results[i]));
            CUDA_CHECK(hipHostFree(h_results[i]));
            CUDA_CHECK(hipStreamDestroy(streams[i]));
        }
        auto delete_end = std::chrono::high_resolution_clock::now();
        delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
    }

    auto end_reps = std::chrono::high_resolution_clock::now();
    double prep_sum = 0, kernel_sum = 0, copy_sum = 0, delete_sum = 0;
    printf("\n===== Timing Summary =====\n");
    for (auto i = 0u; i < prep_times.size(); ++i) {
        if(logReps) {
            printf("Repetition %u:\n", i + 1);
            printf("  Preparation time: %.6f s\n", prep_times[i]);
            printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
            printf("  Data copy time: %.6f s\n", copy_times[i]);
            printf("  Memory deletion time: %.6f s\n", delete_times[i]);
        }
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        copy_sum += copy_times[i];
        delete_sum += delete_times[i];
    }

    auto n = static_cast<double>(prep_times.size());
    double prep_avg = prep_sum / n;
    double kernel_avg = kernel_sum / n;
    double copy_avg = copy_sum / n;
    double delete_avg = delete_sum / n;
    double prep_var = 0, kernel_var = 0, copy_var = 0, delete_var = 0;

    for (auto i = 0u; i < prep_times.size(); ++i) {
        prep_var += (prep_times[i] - prep_avg) * (prep_times[i] - prep_avg);
        kernel_var += (kernel_times[i] - kernel_avg) * (kernel_times[i] - kernel_avg);
        copy_var += (copy_times[i] - copy_avg) * (copy_times[i] - copy_avg);
        delete_var += (delete_times[i] - delete_avg) * (delete_times[i] - delete_avg);
    }

    double prep_std = std::sqrt(prep_var / n);
    double kernel_std = std::sqrt(kernel_var / n);
    double copy_std = std::sqrt(copy_var / n);
    double delete_std = std::sqrt(delete_var / n);

    printf("=========================\n\n");
    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.6f s (stddev: %.6f s)\n", prep_avg, prep_std);
    printf("  Avg kernel execution time: %.6f s (stddev: %.6f s)\n", kernel_avg, kernel_std);
    printf("  Avg data copy time: %.6f s (stddev: %.6f s)\n", copy_avg, copy_std);
    printf("  Avg memory deletion time: %.6f s (stddev: %.6f s)\n", delete_avg, delete_std);
    printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
    printf("=========================\n\n");

    return 0;
}
