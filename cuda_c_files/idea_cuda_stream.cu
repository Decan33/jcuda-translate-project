#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <cstring>
#include "data.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void fourier(
    float tmin,
    float delta,
    int length,
    int coefficients,
    float pi,
    float pi_over_T,
    float result_coefficient,
    float T,
    float *results,
    int stream_offset,
    int stream_size)
{
    auto idx_local = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_local >= stream_size) return;
    auto idx_global = idx_local + stream_offset;
    auto t = tmin + idx_global * delta;
    auto sum = 0.0f;
    for (auto k = 1; k <= coefficients; ++k) {
        auto angle = (2 * k - 1) * pi_over_T * t;
        auto denominator = 4.0f * k * k - 4.0f * k + 1.0f;
        sum += cosf(angle) / denominator;
    }
    results[idx_local] = T * 0.5f - result_coefficient * sum;
}

void performColdRun(float tmin, float tmax, int length, int coefficients, float T, 
                   float delta, float pi, float pi_over_T, float result_coefficient) {
    int chunkSize = (length + NUM_STREAMS - 1) / NUM_STREAMS;
    hipStream_t streams[NUM_STREAMS];
    float* d_results[NUM_STREAMS];
    float* h_results[NUM_STREAMS];

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
        CUDA_CHECK(hipMalloc(&d_results[i], chunkSize * sizeof(float)));
        CUDA_CHECK(hipHostAlloc(&h_results[i], chunkSize * sizeof(float), hipHostMallocDefault));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int startIdx = i * chunkSize;
        int currentChunkSize = std::min(chunkSize, length - startIdx);
        int blocksPerGrid = (currentChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        fourier<<<blocksPerGrid, THREADS_PER_BLOCK, 0, streams[i]>>>(
            tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T,
            d_results[i], startIdx, currentChunkSize
        );
        CUDA_CHECK(hipGetLastError());
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int startIdx = i * chunkSize;
        int currentChunkSize = std::min(chunkSize, length - startIdx);
        CUDA_CHECK(hipMemcpy(h_results[i], d_results[i], currentChunkSize * sizeof(float), hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipFree(d_results[i]));
        CUDA_CHECK(hipHostFree(h_results[i]));
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
}

int main() {
    printf("Performing cold run to warm up GPU...\n");
    performColdRun(tmin, tmax, length, coefficients, T, delta, pi, pi_over_T, result_coefficient);
    printf("Cold run completed.\n\n");

    std::vector<double> prep_times, kernel_times, copy_times, delete_times;
    int chunkSize = (length + NUM_STREAMS - 1) / NUM_STREAMS;
    auto start_reps = std::chrono::high_resolution_clock::now();
    for (auto rep = 0; rep < NUM_REPS; ++rep) {
        auto prep_start = std::chrono::high_resolution_clock::now();
        hipStream_t streams[NUM_STREAMS];
        float* d_results[NUM_STREAMS];
        float* h_results[NUM_STREAMS];
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreate(&streams[i]));
            CUDA_CHECK(hipMalloc(&d_results[i], chunkSize * sizeof(float)));
            CUDA_CHECK(hipHostAlloc(&h_results[i], chunkSize * sizeof(float), hipHostMallocDefault));
        }
        auto prep_end = std::chrono::high_resolution_clock::now();
        prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());
        hipEvent_t kernel_start, kernel_stop;
        CUDA_CHECK(hipEventCreate(&kernel_start));
        CUDA_CHECK(hipEventCreate(&kernel_stop));
        CUDA_CHECK(hipEventRecord(kernel_start));
        for (int i = 0; i < NUM_STREAMS; ++i) {
            int startIdx = i * chunkSize;
            int currentChunkSize = std::min(chunkSize, length - startIdx);
            int blocksPerGrid = (currentChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            fourier<<<blocksPerGrid, THREADS_PER_BLOCK, 0, streams[i]>>>(
                tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T,
                d_results[i], startIdx, currentChunkSize
            );
            CUDA_CHECK(hipGetLastError());
        }
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }

        CUDA_CHECK(hipEventRecord(kernel_stop));
        CUDA_CHECK(hipEventSynchronize(kernel_stop));

        float kernel_ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop));
        kernel_times.push_back(kernel_ms / 1000.0);

        CUDA_CHECK(hipEventDestroy(kernel_start));
        CUDA_CHECK(hipEventDestroy(kernel_stop));
        auto copy_start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < NUM_STREAMS; ++i) {
            int startIdx = i * chunkSize;
            int currentChunkSize = std::min(chunkSize, length - startIdx);
            CUDA_CHECK(hipMemcpy(h_results[i], d_results[i], currentChunkSize * sizeof(float), hipMemcpyDeviceToHost));
        }

        auto copy_end = std::chrono::high_resolution_clock::now();
        copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());
        auto delete_start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < NUM_STREAMS; ++i) {
            CUDA_CHECK(hipFree(d_results[i]));
            CUDA_CHECK(hipHostFree(h_results[i]));
            CUDA_CHECK(hipStreamDestroy(streams[i]));
        }
        auto delete_end = std::chrono::high_resolution_clock::now();
        delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
    }

    auto end_reps = std::chrono::high_resolution_clock::now();
    double prep_sum = 0, kernel_sum = 0, copy_sum = 0, delete_sum = 0;
    printf("\n===== Timing Summary =====\n");
    for (auto i = 0u; i < prep_times.size(); ++i) {
        if(logReps) {
            printf("Repetition %u:\n", i + 1);
            printf("  Preparation time: %.6f s\n", prep_times[i]);
            printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
            printf("  Data copy time: %.6f s\n", copy_times[i]);
            printf("  Memory deletion time: %.6f s\n", delete_times[i]);
        }
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        copy_sum += copy_times[i];
        delete_sum += delete_times[i];
    }

    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.6f s\n", prep_sum / prep_times.size());
    printf("  Avg kernel execution time: %.6f s\n", kernel_sum / kernel_times.size());
    printf("  Avg data copy time: %.6f s\n", copy_sum / copy_times.size());
    printf("  Avg memory deletion time: %.6f s\n", delete_sum / delete_times.size());
    printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
    printf("=========================\n\n");

    return 0;
}
