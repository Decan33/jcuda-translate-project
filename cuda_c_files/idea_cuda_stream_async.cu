#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <fstream>
#include <vector>
#include <chrono>
#include "data.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

constexpr int CHUNK_SIZE = 10000000;

__constant__ float d_coefficients[MAX_COEFFICIENTS];
__constant__ float d_params[5];  // [tmin, tmax, length, coefficients, delta]

void initConstantMemory(int coefficients, float tmin, float tmax, int length, float delta) {
    if (coefficients > MAX_COEFFICIENTS) {
        std::cerr << "Error: coefficients > " << MAX_COEFFICIENTS << " (" << coefficients << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
    float h_coefficients[MAX_COEFFICIENTS];
    for (int k = 1; k <= coefficients; ++k) {
        h_coefficients[k - 1] = 1.0f / (4.0f * k * k - 4.0f * k + 1.0f);
    }
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_coefficients), h_coefficients, coefficients * sizeof(float)));
    float h_params[5] = {tmin, tmax, static_cast<float>(length), static_cast<float>(coefficients), delta};
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_params), h_params, 5 * sizeof(float)));
}

__global__ void fourier(int start_idx, int end_idx, float *results)
{
    auto idx = start_idx + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= end_idx) return;
    auto t = d_params[0] + idx * d_params[4];
    auto sum = 0.0f;
    constexpr float pi = 3.14159265f;
    auto coeff = static_cast<int>(d_params[3]);
    if (coeff > MAX_COEFFICIENTS) coeff = MAX_COEFFICIENTS;

    extern __shared__ float s_angles[];

    for (auto k = 1; k <= coeff; ++k) {
        s_angles[k - 1] = (2 * k - 1) * pi * t;
    }
    __syncthreads();
    for (auto k = 1; k <= coeff; ++k) {
        sum += cosf(s_angles[k - 1]) * d_coefficients[k - 1];
    }
    results[idx - start_idx] = 0.5f - (4.0f * sum) / (pi * pi);
}

void performColdRun(float tmin, float tmax, int length, int coefficients, float delta) {
    initConstantMemory(coefficients, tmin, tmax, length, delta);
    float *d_results;
    CUDA_CHECK(hipMalloc(&d_results, CHUNK_SIZE * sizeof(float)));
    float *h_results;
    CUDA_CHECK(hipHostAlloc(&h_results, CHUNK_SIZE * sizeof(float), hipHostMallocDefault | hipHostMallocMapped));
    auto sharedMemSize = coefficients * sizeof(float);

    for (auto chunkStart = 0; chunkStart < length; chunkStart += CHUNK_SIZE) {
        auto chunkEnd = std::min(chunkStart + CHUNK_SIZE, length);
        auto thisChunkSize = chunkEnd - chunkStart;
        auto blocks = (thisChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        fourier<<<blocks, THREADS_PER_BLOCK, sharedMemSize>>>(chunkStart, chunkEnd, d_results);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(h_results, d_results, thisChunkSize * sizeof(float), hipMemcpyDeviceToHost));
    }

    CUDA_CHECK(hipFree(d_results));
    CUDA_CHECK(hipHostFree(h_results));
}

int main()
{
    printf("Performing cold run to warm up GPU...\n");
    performColdRun(tmin, tmax, length, coefficients, delta);
    printf("Cold run completed.\n\n");
    
    std::vector<double> prep_times, kernel_times, delete_times;

    auto start_reps = std::chrono::high_resolution_clock::now();
    for (auto rep = 0; rep < NUM_REPS; rep++) {
        auto prep_start = std::chrono::high_resolution_clock::now();
        
        initConstantMemory(coefficients, tmin, tmax, length, delta);
        
        float *d_results;
        CUDA_CHECK(hipMalloc(&d_results, CHUNK_SIZE * sizeof(float)));
        
        float *h_results;
        CUDA_CHECK(hipHostAlloc(&h_results, CHUNK_SIZE * sizeof(float), hipHostMallocDefault | hipHostMallocMapped));
        
        auto sharedMemSize = coefficients * sizeof(float);
        
        auto prep_end = std::chrono::high_resolution_clock::now();
        prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());

        hipEvent_t kernel_start, kernel_stop;
        CUDA_CHECK(hipEventCreate(&kernel_start));
        CUDA_CHECK(hipEventCreate(&kernel_stop));
        CUDA_CHECK(hipEventRecord(kernel_start));

        for (auto chunkStart = 0; chunkStart < length; chunkStart += CHUNK_SIZE) {
            auto chunkEnd = std::min(chunkStart + CHUNK_SIZE, length);
            auto thisChunkSize = chunkEnd - chunkStart;
            auto blocks = (thisChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

            fourier<<<blocks, THREADS_PER_BLOCK, sharedMemSize>>>(chunkStart, chunkEnd, d_results);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());

            CUDA_CHECK(hipMemcpy(h_results, d_results, thisChunkSize * sizeof(float), hipMemcpyDeviceToHost));
        }
        
        CUDA_CHECK(hipEventRecord(kernel_stop));
        CUDA_CHECK(hipEventSynchronize(kernel_stop));
        
        float kernel_ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop));
        kernel_times.push_back(kernel_ms / 1000.0);
        
        CUDA_CHECK(hipEventDestroy(kernel_start));
        CUDA_CHECK(hipEventDestroy(kernel_stop));

        auto delete_start = std::chrono::high_resolution_clock::now();
        
        CUDA_CHECK(hipFree(d_results));
        CUDA_CHECK(hipHostFree(h_results));
        
        auto delete_end = std::chrono::high_resolution_clock::now();
        delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
    }
    auto end_reps = std::chrono::high_resolution_clock::now();

    double prep_sum = 0, kernel_sum = 0, delete_sum = 0;
    printf("\n===== Timing Summary =====\n");
    
    for (auto i = 0u; i < prep_times.size(); ++i) {
	    if(logReps) {
	        printf("Repetition %u:\n", i + 1);
        	printf("  Preparation time: %.6f s\n", prep_times[i]);
        	printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
        	printf("  Data copy time: %.6f s\n", copy_times[i]);
        	printf("  Memory deletion time: %.6f s\n", delete_times[i]);
	    }
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        delete_sum += delete_times[i];
    }
    
    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.6f s\n", prep_sum / prep_times.size());
    printf("  Avg kernel execution time: %.6f s\n", kernel_sum / kernel_times.size());
    printf("  Avg memory deletion time: %.6f s\n", delete_sum / delete_times.size());
    printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
    printf("=========================\n\n");
    return 0;
}