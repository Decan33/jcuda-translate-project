#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <chrono>
#include "data.h"

__global__ void fourier(
    float tmin,
    float delta,
    int length,
    int coefficients,
    float pi,
    float pi_over_T,
    float result_coefficient,
    float T,
    float *results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length) return;
    float t = tmin + idx * delta;
    float sum = 0.0f;
    for (int k = 1; k <= coefficients; ++k) {
        float angle = (2 * k - 1) * pi_over_T * t;
        float denominator = 4.0f * k * k - 4.0f * k + 1.0f;
        sum += cosf(angle) / denominator;
    }
    results[idx] = T * 0.5f - result_coefficient * sum;
}


static float *d_results = nullptr;
static float *h_results = nullptr;
static hipEvent_t kernel_start, kernel_stop;
static bool memory_initialized = false;

void initializeMemory() {
    if (memory_initialized) return;

    CUDA_CHECK(hipMalloc(&d_results, length * sizeof(float)));
    h_results = new float[length];
    CUDA_CHECK(hipEventCreate(&kernel_start));
    CUDA_CHECK(hipEventCreate(&kernel_stop));

    memory_initialized = true;
}

void cleanupMemory() {
    if (!memory_initialized) return;

    CUDA_CHECK(hipFree(d_results));
    delete[] h_results;
    CUDA_CHECK(hipEventDestroy(kernel_start));
    CUDA_CHECK(hipEventDestroy(kernel_stop));

    memory_initialized = false;
}

void performColdRun() {
    printf("Performing cold run to warm up GPU...\n");

    initializeMemory();

    fourier<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(
        tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T, d_results
    );
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Cold run completed.\n\n");
}

void runSingleTest(std::vector<double>& prep_times, std::vector<double>& kernel_times,
                   std::vector<double>& copy_times, std::vector<double>& cleanup_times) {


    auto prep_start = std::chrono::high_resolution_clock::now();
    auto prep_end = std::chrono::high_resolution_clock::now();
    prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());

    CUDA_CHECK(hipEventRecord(kernel_start));
    fourier<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(
        tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T, d_results
    );
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(kernel_stop));
    CUDA_CHECK(hipEventSynchronize(kernel_stop));

    float kernel_ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop));
    kernel_times.push_back(kernel_ms / 1000.0);


    auto copy_start = std::chrono::high_resolution_clock::now();
    CUDA_CHECK(hipMemcpy(h_results, d_results, length * sizeof(float), hipMemcpyDeviceToHost));
    auto copy_end = std::chrono::high_resolution_clock::now();
    copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());

    auto cleanup_start = std::chrono::high_resolution_clock::now();

    auto cleanup_end = std::chrono::high_resolution_clock::now();
    cleanup_times.push_back(std::chrono::duration<double>(cleanup_end - cleanup_start).count());
}

void printResults(const std::vector<double>& prep_times, const std::vector<double>& kernel_times,
                  const std::vector<double>& copy_times, const std::vector<double>& cleanup_times,
                  double total_time) {

    if (logReps) {
        for (size_t i = 0; i < prep_times.size(); ++i) {
            printf("Repetition %zu:\n", i + 1);
            printf("  Preparation time: %.6f s\n", prep_times[i]);
            printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
            printf("  Data copy time: %.6f s\n", copy_times[i]);
            printf("  Memory cleanup time: %.6f s\n", cleanup_times[i]);
        }
    }

    double prep_sum = 0, kernel_sum = 0, copy_sum = 0, cleanup_sum = 0;
    for (size_t i = 0; i < prep_times.size(); ++i) {
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        copy_sum += copy_times[i];
        cleanup_sum += cleanup_times[i];
    }

    const double n = static_cast<double>(prep_times.size());
    const double prep_avg = prep_sum / n;
    const double kernel_avg = kernel_sum / n;
    const double copy_avg = copy_sum / n;
    const double cleanup_avg = cleanup_sum / n;

    double prep_var = 0, kernel_var = 0, copy_var = 0, cleanup_var = 0;
    for (size_t i = 0; i < prep_times.size(); ++i) {
        prep_var += (prep_times[i] - prep_avg) * (prep_times[i] - prep_avg);
        kernel_var += (kernel_times[i] - kernel_avg) * (kernel_times[i] - kernel_avg);
        copy_var += (copy_times[i] - copy_avg) * (copy_times[i] - copy_avg);
        cleanup_var += (cleanup_times[i] - cleanup_avg) * (cleanup_times[i] - cleanup_avg);
    }

    const double prep_std = std::sqrt(prep_var / n);
    const double kernel_std = std::sqrt(kernel_var / n);
    const double copy_std = std::sqrt(copy_var / n);
    const double cleanup_std = std::sqrt(cleanup_var / n);

    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.3f s (stddev: %.3f s)\n", prep_avg, prep_std);
    printf("  Avg kernel execution time: %.3f s (stddev: %.3f s)\n", kernel_avg, kernel_std);
    printf("  Avg data copy time: %.3f s (stddev: %.3f s)\n", copy_avg, copy_std);
    printf("  Avg memory cleanup time: %.3f s (stddev: %.3f s)\n", cleanup_avg, cleanup_std);
    printf("  Whole time taken for %d reps: %.3f s\n", NUM_REPS, total_time);
    printf("=========================\n\n");
}

int main() {
    performColdRun();

    std::vector<double> prep_times, kernel_times, copy_times, cleanup_times;
    prep_times.reserve(NUM_REPS);
    kernel_times.reserve(NUM_REPS);
    copy_times.reserve(NUM_REPS);
    cleanup_times.reserve(NUM_REPS);

    auto start_reps = std::chrono::high_resolution_clock::now();
    for (int rep = 0; rep < NUM_REPS; ++rep) {
        runSingleTest(prep_times, kernel_times, copy_times, cleanup_times);
    }
    auto end_reps = std::chrono::high_resolution_clock::now();

    const double total_time = std::chrono::duration<double>(end_reps - start_reps).count();

    printResults(prep_times, kernel_times, copy_times, cleanup_times, total_time);

    cleanupMemory();
    return 0;
}