#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <chrono>
#include "data.h"

__global__ void fourier(
    float tmin,
    float delta,
    int length,
    int coefficients,
    float pi,
    float pi_over_T,
    float result_coefficient,
    float T,
    float *results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length) return;
    float t = tmin + idx * delta;
    float sum = 0.0f;
    for (int k = 1; k <= coefficients; ++k) {
        float angle = (2 * k - 1) * pi_over_T * t;
        float denominator = 4.0f * k * k - 4.0f * k + 1.0f;
        sum += cosf(angle) / denominator;
    }
    results[idx] = T * 0.5f - result_coefficient * sum;
}

void performColdRun(float tmin, float delta, int length, int coefficients, 
                   float pi, float pi_over_T, float result_coefficient, float T) {
    float *d_results;
    hipMalloc(&d_results, length * sizeof(float));

    auto threadsPerBlock = THREADS_PER_BLOCK;
    auto blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

    fourier<<<blocksPerGrid, threadsPerBlock>>>(
        tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T, d_results
    );
    hipDeviceSynchronize();

    float *h_results = new float[length];
    hipMemcpy(h_results, d_results, length * sizeof(float), hipMemcpyDeviceToHost);

    delete[] h_results;
    hipFree(d_results);
}

int main() {
    printf("Performing cold run to warm up GPU...\n");
    performColdRun(tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T);
    printf("Cold run completed.\n\n");

    std::vector<double> prep_times, kernel_times, copy_times, delete_times;

    auto start_reps = std::chrono::high_resolution_clock::now();
    for (auto rep = 0; rep < NUM_REPS; ++rep) {
        auto prep_start = std::chrono::high_resolution_clock::now();
        
        float *d_results;
        hipMalloc(&d_results, length * sizeof(float));

        auto threadsPerBlock = THREADS_PER_BLOCK;
        auto blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
        
        auto prep_end = std::chrono::high_resolution_clock::now();
        prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());

        hipEvent_t kernel_start, kernel_stop;
        hipEventCreate(&kernel_start);
        hipEventCreate(&kernel_stop);
        hipEventRecord(kernel_start);

        fourier<<<blocksPerGrid, threadsPerBlock>>>(
            tmin, delta, length, coefficients, pi, pi_over_T, result_coefficient, T, d_results
        );
        hipDeviceSynchronize();
        hipEventRecord(kernel_stop);
        hipEventSynchronize(kernel_stop);
        
        float kernel_ms = 0.0f;
        hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop);

        kernel_times.push_back(kernel_ms / 1000.0);

        hipEventDestroy(kernel_start);
        hipEventDestroy(kernel_stop);

        auto copy_start = std::chrono::high_resolution_clock::now();
        float *h_results = new float[length];
        hipMemcpy(h_results, d_results, length * sizeof(float), hipMemcpyDeviceToHost);
        auto copy_end = std::chrono::high_resolution_clock::now();
        copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());

        auto delete_start = std::chrono::high_resolution_clock::now();
        delete[] h_results;
        hipFree(d_results);
        auto delete_end = std::chrono::high_resolution_clock::now();

        delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
    }
    auto end_reps = std::chrono::high_resolution_clock::now();
    
    double prep_sum = 0, kernel_sum = 0, copy_sum = 0, delete_sum = 0;
    
    for (auto i = 0u; i < prep_times.size(); ++i) {
	    if(logReps) {
	        printf("Repetition %u:\n", i + 1);
        	printf("  Preparation time: %.6f s\n", prep_times[i]);
        	printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
        	printf("  Data copy time: %.6f s\n", copy_times[i]);
        	printf("  Memory deletion time: %.6f s\n", delete_times[i]);
	    }
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        copy_sum += copy_times[i];
        delete_sum += delete_times[i];
    }

    auto n = static_cast<double>(prep_times.size());
    double prep_avg = prep_sum / n;
    double kernel_avg = kernel_sum / n;
    double copy_avg = copy_sum / n;
    double delete_avg = delete_sum / n;
    double prep_var = 0, kernel_var = 0, copy_var = 0, delete_var = 0;
    
    for (auto i = 0u; i < prep_times.size(); ++i) {
        prep_var += (prep_times[i] - prep_avg) * (prep_times[i] - prep_avg);
        kernel_var += (kernel_times[i] - kernel_avg) * (kernel_times[i] - kernel_avg);
        copy_var += (copy_times[i] - copy_avg) * (copy_times[i] - copy_avg);
        delete_var += (delete_times[i] - delete_avg) * (delete_times[i] - delete_avg);
    }

    double prep_std = std::sqrt(prep_var / n);
    double kernel_std = std::sqrt(kernel_var / n);
    double copy_std = std::sqrt(copy_var / n);
    double delete_std = std::sqrt(delete_var / n);
    
    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.6f s (stddev: %.6f s)\n", prep_avg, prep_std);
    printf("  Avg kernel execution time: %.6f s (stddev: %.6f s)\n", kernel_avg, kernel_std);
    printf("  Avg data copy time: %.6f s (stddev: %.6f s)\n", copy_avg, copy_std);
    printf("  Avg memory deletion time: %.6f s (stddev: %.6f s)\n", delete_avg, delete_std);
    printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
    printf("=========================\n\n");
    return 0;
}
