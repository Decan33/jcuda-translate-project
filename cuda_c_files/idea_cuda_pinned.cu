#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <chrono>
#include <cstring>
#include "data.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

constexpr int CHUNK_SIZE = 250000000; // match Java

__constant__ float const_tmin;
__constant__ float const_delta;
__constant__ int   const_coefficients;

__constant__ float const_pi;
__constant__ float const_pi_squared;
__constant__ float const_T;
__constant__ float const_pi_over_T;
__constant__ float constant_result_coefficient;

__global__ void fourier(float* results)
{
    __shared__ float shared_coefficients[MAX_COEFFICIENTS];

    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    auto idx = threadIdx.x;

    for (auto k = idx; k < const_coefficients; k += blockDim.x)
    {
        auto denominator = 4.0f * (k + 1) * (k + 1) - 4.0f * (k + 1) + 1.0f;
        shared_coefficients[k] = 1.0f / denominator;
    }

    __syncthreads();

    if (tid >= gridDim.x * blockDim.x) return;

    auto t = const_tmin + tid * const_delta;
    auto sum = 0.0f;

    for (auto k = 1; k <= const_coefficients; ++k)
    {
        auto angle = (2 * k - 1) * const_pi_over_T * t;
        auto numerator = cosf(angle);
        sum += numerator * shared_coefficients[k - 1];
    }

    results[tid] = const_T * 0.5f - (constant_result_coefficient * sum);
}

void performColdRun(float tmin, float tmax, int length, int coefficients, float delta) {
    float* result_device;
    CUDA_CHECK(hipMalloc((void**)&result_device, length * sizeof(float)));

    auto blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_tmin),   &tmin, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_delta),  &delta, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_coefficients), &coefficients, sizeof(int)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi), &pi, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi_squared), &pi_squared, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_T), &T, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi_over_T), &pi_over_T, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constant_result_coefficient), &result_coefficient, sizeof(float)));

    fourier<<<blocks, THREADS_PER_BLOCK>>>(result_device);
    CUDA_CHECK(hipDeviceSynchronize());

    int actualChunkSize = std::min(CHUNK_SIZE, length);
    float* result_host;
    CUDA_CHECK(hipHostAlloc((void**)&result_host, actualChunkSize * sizeof(float), hipHostMallocDefault));
    for (int offset = 0; offset < length; offset += actualChunkSize) {
        int thisChunk = std::min(actualChunkSize, length - offset);
        CUDA_CHECK(hipMemcpy(result_host, result_device + offset, thisChunk * sizeof(float), hipMemcpyDeviceToHost));
    }
    CUDA_CHECK(hipHostFree(result_host));
    CUDA_CHECK(hipFree(result_device));
}

int main()
{
    long long requiredDeviceMemory = (long long)length * sizeof(float);
    long long requiredHostMemory = (long long)std::min(CHUNK_SIZE, length) * sizeof(float);
    printf("Memory requirements: Device=%.2f GB, Host chunk=%.2f MB\n",
        requiredDeviceMemory / (1024.0 * 1024.0 * 1024.0),
        requiredHostMemory / (1024.0 * 1024.0));
    printf("TESTING FOURIER USING PINNED MEMORY\n");
    printf("Performing cold run to warm up GPU...\n");
    performColdRun(tmin, tmax, length, coefficients, delta);
    printf("Cold run completed.\n\n");

    std::vector<double> prep_times, kernel_times, copy_times, delete_times;

    auto start_reps = std::chrono::high_resolution_clock::now();
    for (auto rep = 0; rep < NUM_REPS; ++rep) {
        auto prep_start = std::chrono::high_resolution_clock::now();
        
        float* result_device;
        CUDA_CHECK(hipMalloc((void**)&result_device, length * sizeof(float)));

        auto blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_tmin),   &tmin, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_delta),  &delta, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_coefficients), &coefficients, sizeof(int)));

        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi), &pi, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi_squared), &pi_squared, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_T), &T, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_pi_over_T), &pi_over_T, sizeof(float)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constant_result_coefficient), &result_coefficient, sizeof(float)));

        auto prep_end = std::chrono::high_resolution_clock::now();
        prep_times.push_back(std::chrono::duration<double>(prep_end - prep_start).count());
        
        hipEvent_t kernel_start, kernel_stop;
        CUDA_CHECK(hipEventCreate(&kernel_start));
        CUDA_CHECK(hipEventCreate(&kernel_stop));
        CUDA_CHECK(hipEventRecord(kernel_start));
        
        fourier<<<blocks, THREADS_PER_BLOCK>>>(result_device);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipEventRecord(kernel_stop));
        CUDA_CHECK(hipEventSynchronize(kernel_stop));
        
        float kernel_ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&kernel_ms, kernel_start, kernel_stop));
        kernel_times.push_back(kernel_ms / 1000.0);
        
        CUDA_CHECK(hipEventDestroy(kernel_start));
        CUDA_CHECK(hipEventDestroy(kernel_stop));
        
        auto copy_start = std::chrono::high_resolution_clock::now();
        int actualChunkSize = std::min(CHUNK_SIZE, length);
        float* result_host;
        CUDA_CHECK(hipHostAlloc((void**)&result_host, actualChunkSize * sizeof(float), hipHostMallocDefault));
        for (int offset = 0; offset < length; offset += actualChunkSize) {
            int thisChunk = std::min(actualChunkSize, length - offset);
            CUDA_CHECK(hipMemcpy(result_host, result_device + offset, thisChunk * sizeof(float), hipMemcpyDeviceToHost));
        }
        auto copy_end = std::chrono::high_resolution_clock::now();
        copy_times.push_back(std::chrono::duration<double>(copy_end - copy_start).count());
        
        auto delete_start = std::chrono::high_resolution_clock::now();
        CUDA_CHECK(hipHostFree(result_host));
        CUDA_CHECK(hipFree(result_device));
        auto delete_end = std::chrono::high_resolution_clock::now();
        
        delete_times.push_back(std::chrono::duration<double>(delete_end - delete_start).count());
    }
    auto end_reps = std::chrono::high_resolution_clock::now();
    
    double prep_sum = 0, kernel_sum = 0, copy_sum = 0, delete_sum = 0;
    
    for (auto i = 0u; i < prep_times.size(); ++i) {
	    if(logReps) {
	        printf("Repetition %u:\n", i + 1);
        	printf("  Preparation time: %.6f s\n", prep_times[i]);
        	printf("  Kernel execution time: %.6f s\n", kernel_times[i]);
        	printf("  Data copy time: %.6f s\n", copy_times[i]);
        	printf("  Memory deletion time: %.6f s\n", delete_times[i]);
	    }
        prep_sum += prep_times[i];
        kernel_sum += kernel_times[i];
        copy_sum += copy_times[i];
        delete_sum += delete_times[i];
    }
    
    auto n = static_cast<double>(prep_times.size());
    double prep_avg = prep_sum / n;
    double kernel_avg = kernel_sum / n;
    double copy_avg = copy_sum / n;
    double delete_avg = delete_sum / n;
    double prep_var = 0, kernel_var = 0, copy_var = 0, delete_var = 0;
    
    for (auto i = 0u; i < prep_times.size(); ++i) {
        prep_var += (prep_times[i] - prep_avg) * (prep_times[i] - prep_avg);
        kernel_var += (kernel_times[i] - kernel_avg) * (kernel_times[i] - kernel_avg);
        copy_var += (copy_times[i] - copy_avg) * (copy_times[i] - copy_avg);
        delete_var += (delete_times[i] - delete_avg) * (delete_times[i] - delete_avg);
    }
    
    double prep_std = std::sqrt(prep_var / n);
    double kernel_std = std::sqrt(kernel_var / n);
    double copy_std = std::sqrt(copy_var / n);
    double delete_std = std::sqrt(delete_var / n);
    
    printf("\nAverages over %zu repetitions:\n", prep_times.size());
    printf("  Avg preparation time: %.6f s (stddev: %.6f s)\n", prep_avg, prep_std);
    printf("  Avg kernel execution time: %.6f s (stddev: %.6f s)\n", kernel_avg, kernel_std);
    printf("  Avg data copy time: %.6f s (stddev: %.6f s)\n", copy_avg, copy_std);
    printf("  Avg memory deletion time: %.6f s (stddev: %.6f s)\n", delete_avg, delete_std);
    printf("  Whole time taken for %d reps: %.6f s\n", NUM_REPS, std::chrono::duration<double>(end_reps - start_reps).count());
    printf("=========================\n\n");
    return 0;
} 